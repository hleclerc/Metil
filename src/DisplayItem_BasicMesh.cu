#include "hip/hip_runtime.h"
#include "GetSurroundingBox_BasicMesh.h"
#include "DisplayItem_BasicMesh.h"
#include "BitmapDisplay.h"
#include "MathBasicVec.h"
#include "CudaMetil.h"

#define NB_PIX_RASTER_BOX 32
#define NB_BLOCKS_FOR_ELEM_COUNT 32
#define MAX_WH ( 1600 * 1200 )
#define NB_THREADS_FOR_RASTER 64
#define NB_THREADS_FOR_PROJ 64

BEG_METIL_NAMESPACE;

typedef DisplayTrans::T3 T3;
typedef DisplayTrans::T T;

DisplayItem_BasicMesh::DisplayItem_BasicMesh( Ps<BasicMesh_Compacted> mesh ) : mesh( mesh ) {
    for( int d = 0; d < 3; ++d )
        proj[ d ] = 0;

    size_elem_count = 0;
    elem_count_gpu = 0;

    size_elem_data = 0;
    elem_data_gpu = 0;

    num_coloring_field = -1;
    dim_coloring_field = 0;
}

DisplayItem_BasicMesh::~DisplayItem_BasicMesh() {
    for( int d = 0; d < 3; ++d )
        hipFree( proj[ d ] );
    hipFree( elem_count_gpu );
    hipFree( elem_data_gpu  );
}

__inline__ ST rese_elem_count_for_one_group( int sb ) {
    return ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1 /*offset list needs n + 1 terms...*/;
}

unsigned *DisplayItem_BasicMesh::get_elem_count_gpu_ptr( ST nb_types, int sb ) {
    int needed_size = nb_types * rese_elem_count_for_one_group( sb ) + 1 /*size for total nb elements*/;
    if ( size_elem_count < needed_size ) {
        if ( elem_count_gpu )
            hipFree( elem_count_gpu );
        CSC( hipMalloc( &elem_count_gpu, sizeof( unsigned ) * needed_size ) );
        size_elem_count = needed_size;
    }
    return elem_count_gpu;
}

int *DisplayItem_BasicMesh::get_elem_data_gpu_ptr( ST needed_size ) {
    if ( size_elem_data < needed_size ) {
        needed_size = needed_size * 8 / 7;
        if ( elem_data_gpu )
            hipFree( elem_data_gpu );
        CSC( hipMalloc( &elem_data_gpu, sizeof( int ) * needed_size ) );
        size_elem_data = needed_size;
    }
    return elem_data_gpu;
}

void DisplayItem_BasicMesh::update_p_min_p_max( GenericDisplay *display, T3 &p_min, T3 &p_max, bool use_trans ) {
    T3 t_min;
    T3 t_max;
    get_surrounding_box( t_min.ptr(), t_max.ptr(), mesh.ptr(), use_trans ? display->get_trans_gpu() : 0, display->get_w(), display->get_h() );
    for( int i = 0; i < 3; ++i ) {
        p_min[ i ] = min( p_min[ i ], t_min[ i ] );
        p_max[ i ] = max( p_max[ i ], t_max[ i ] );
    }
}

void DisplayItem_BasicMesh::set_coloring_field( int n, T mi, T ma, int dimension ) {
    num_coloring_field = n;
    dim_coloring_field = dimension;
    min_coloring_field = mi;
    max_coloring_field = ma;
}


__global__
void make_proj_kernel( T *x, T *y, T *z, const DisplayTrans *trans_ptr, int w, int h, BasicMesh_Compacted *m ) {
    DisplayTrans::Buf trans = trans_ptr->make_buf( w, h );

    int n = blockIdx.x * NB_THREADS_FOR_PROJ + threadIdx.x;
    if ( n < m->pos_nodes[ 0 ].size() ) {
        const T *n_x = m->pos_nodes[ 0 ].ptr();
        const T *n_y = m->pos_nodes[ 1 ].ptr();
        const T *n_z = m->pos_nodes[ 2 ].ptr();

        T3 B = trans.proj( T3( n_x[ n ], n_y[ n ], n_z[ n ] ) );

        x[ n ] = round( B[ 0 ] );
        y[ n ] = round( B[ 1 ] );
        z[ n ] = B[ 2 ];
    }
}

__global__
void make_elem_count_kernel( unsigned *elem_count, int wb, int hb, int sb, const BasicMesh_Compacted *m, const T *n_x, const T *n_y ) {
    elem_count += blockIdx.x * sb;

    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        __shared__ unsigned local_count[ MAX_WH / NB_PIX_RASTER_BOX / NB_PIX_RASTER_BOX ];
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            local_count[ i ] = 0;
        __syncthreads(); //wait until local_count is ok

        int nb_elem = m->elem_groups[ num_type ].connec[ 0 ].size();
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        int be = ( blockIdx.x + 0 ) * nb_elem / gridDim.x;
        int ee = ( blockIdx.x + 1 ) * nb_elem / gridDim.x;

        for( int i = be + threadIdx.x; i < ee; i += blockDim.x ) {
            int n_0 = c_0[ i ];
            int n_1 = c_1[ i ];
            int n_2 = c_2[ i ];

            T x_0 = n_x[ n_0 ], y_0 = n_y[ n_0 ];
            T x_1 = n_x[ n_1 ], y_1 = n_y[ n_1 ];
            T x_2 = n_x[ n_2 ], y_2 = n_y[ n_2 ];

            int x_m = int( min( min( x_0, x_1 ), x_2 ) ) / NB_PIX_RASTER_BOX;
            int y_m = int( min( min( y_0, y_1 ), y_2 ) ) / NB_PIX_RASTER_BOX;

            int x_M = int( max( max( x_0, x_1 ), x_2 ) ) / NB_PIX_RASTER_BOX;
            int y_M = int( max( max( y_0, y_1 ), y_2 ) ) / NB_PIX_RASTER_BOX;

            for( int yb = max( 0, y_m ); yb <= min( hb - 1, y_M ); ++yb )
                for( int xb = max( 0, x_m ); xb <= min( wb - 1, x_M ); ++xb )
                    atomicAdd( local_count + wb * yb + xb, 1 );
        }

        // copy local count in global memory
        __syncthreads();
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            elem_count[ i ] = local_count[ i ];
    }
}


__global__
void make_sum_elem_count_kernel( unsigned *elem_count, int wb, int hb, int nb_types ) {
    int sb = wb * hb;
    for( int num_type = 0; num_type < nb_types; ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        int n = blockIdx.x * blockDim.x + threadIdx.x;
        if ( n < sb ) {
            unsigned acc = 0;
            for( int i = n; i < sb * NB_BLOCKS_FOR_ELEM_COUNT; i += sb ) {
                unsigned val = elem_count[ i ];
                elem_count[ i ] = acc;
                acc += val;
            }
            elem_count[ sb * NB_BLOCKS_FOR_ELEM_COUNT + n ] = acc;
        }
    }
}

// elem_count will be a list of offset. total number of elements is stored at the end of elem_count
__global__
void make_off_elem_count_kernel( unsigned *elem_count, int wb, int hb, int nb_types ) {
    int sb = wb * hb;
    int offset = sb * NB_BLOCKS_FOR_ELEM_COUNT;

    unsigned acc = 0;
    for( int num_type = 0; num_type < nb_types; ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        for( int i = 0; i < sb; ++i ) {
            unsigned val = elem_count[ offset + i ];
            elem_count[ offset + i ] = acc;
            acc += val;
        }
        elem_count[ offset + sb ] = acc;
    }

    *elem_count = acc;
}

///
__global__
void make_elem_data_kernel( unsigned *elem_count, int *elem_data, const DisplayTrans *trans_ptr, int wb, int hb, int sb, BasicMesh_Compacted *m, const T *n_x, const T *n_y ) {
    __shared__ unsigned local_count[ MAX_WH / NB_PIX_RASTER_BOX / NB_PIX_RASTER_BOX ];

    // int size_for_1t = ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1;
    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        __syncthreads();
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            local_count[ i ] = elem_count[ NB_BLOCKS_FOR_ELEM_COUNT * sb + i ] + elem_count[ blockIdx.x * sb + i ];

        int nb_elem = m->elem_groups[ num_type ].connec[ 0 ].size();
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        int be = ( blockIdx.x + 0 ) * nb_elem / gridDim.x;
        int ee = ( blockIdx.x + 1 ) * nb_elem / gridDim.x;

        __syncthreads();
        for( int i = be + threadIdx.x; i < ee; i += blockDim.x ) {
            int n_0 = c_0[ i ];
            int n_1 = c_1[ i ];
            int n_2 = c_2[ i ];

            T x_0 = n_x[ n_0 ], y_0 = n_y[ n_0 ];
            T x_1 = n_x[ n_1 ], y_1 = n_y[ n_1 ];
            T x_2 = n_x[ n_2 ], y_2 = n_y[ n_2 ];

            int x_m = int( min( min( x_0, x_1 ), x_2 ) ) / NB_PIX_RASTER_BOX;
            int y_m = int( min( min( y_0, y_1 ), y_2 ) ) / NB_PIX_RASTER_BOX;

            int x_M = int( max( max( x_0, x_1 ), x_2 ) ) / NB_PIX_RASTER_BOX;
            int y_M = int( max( max( y_0, y_1 ), y_2 ) ) / NB_PIX_RASTER_BOX;

            for( int yb = max( 0, y_m ); yb <= min( hb - 1, y_M ); ++yb ) {
                for( int xb = max( 0, x_m ); xb <= min( wb - 1, x_M ); ++xb ) {
                    unsigned index = atomicAdd( local_count + wb * yb + xb, 1 );
                    elem_data[ index + 0 ] = i;
                }
            }
        }
    }
}

__global__
void DisplayItem_BasicMesh_render_kernel( unsigned *img, const unsigned *elem_count, int w, int h, int wb, int hb ) {
    int sb = wb * hb;
    elem_count += sb;
    for( int y = 0; y < h; ++y )
        for( int x = 0; x < w; ++x )
            img[ y * w + x ] = 0xFF000000 + elem_count[ y / NB_PIX_RASTER_BOX * wb + x / NB_PIX_RASTER_BOX ] * 25;
}

__inline__
unsigned shader( float z_n, float c ) {
    if ( z_n < 0 or z_n > 1 ) {
        int r = 200 * c;
        int g = r;
        int b = r;
        int a = 255;
        return ( r << 0 ) + ( g << 8 ) + ( b << 16 ) + ( a << 24 );
    }
    int r = ( z_n * 255 ) * c;
    int g = ( 0         ) * c;
    int b = ( 255 - r   ) * c;
    int a = 255;
    return ( r << 0 ) + ( g << 8 ) + ( b << 16 ) + ( a << 24 );
}

///// use 2 unsigned to store a T ( 2 because of tha alpha chanel that must be == 0xFF )
//__inline__ void copy_T_as_I( unsigned *res, float src ) {
//    int e, s = src < 0;
//    unsigned m = abs( frexpf( src, &e ) * 0x01000000 );
//    res[ 0 ] = 0xFF000000 + m;
//    res[ 1 ] = 0xFF000000 + ( s << 16 ) + ( e + 32768 );
//}

__global__
void raster_gpu_kernel( unsigned *rgba, unsigned *zznv, unsigned *nnnn, int wb, int hb, int sb, int w, int h, const unsigned *elem_offsets, const int *elem_data, const BasicMesh_Compacted *m, const T *n_x, const T *n_y, const T *n_z, float z_min, float z_mul, bool first_item,
                       float min_coloring_field, float max_coloring_field, int num_coloring_field, int dim_coloring_field ) {
    int bx = blockIdx.x * NB_PIX_RASTER_BOX;
    int by = blockIdx.y * NB_PIX_RASTER_BOX;

    // initialization of the z buffer
    __shared__ unsigned rgba_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    __shared__ unsigned zznv_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    __shared__ unsigned nnnn_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    if ( first_item ) {
        for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
            rgba_buffer[ i ] = 0x00000000;
            zznv_buffer[ i ] = 0xFFFFFFFF;
            nnnn_buffer[ i ] = 0xFFFFFFFF;
        }
    } else {
        for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
            int x = bx + i % NB_PIX_RASTER_BOX;
            int y = by + i / NB_PIX_RASTER_BOX;
            if ( x < w and y < h ) {
                rgba_buffer[ i ] = rgba[ w * y + x ];
                zznv_buffer[ i ] = zznv[ w * y + x ];
                nnnn_buffer[ i ] = nnnn[ w * y + x ];
            }
        }
    }

    const float *n_v = num_coloring_field >= 0 ? m->nodal_fields[ num_coloring_field ].data[ dim_coloring_field ].ptr() : 0;

    int size_for_1t = ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1;
    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_offsets += size_for_1t ) {
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        // swwep elem list
        __syncthreads(); //wait until z_buffer is ok
        unsigned beg_elem = elem_offsets[ wb * blockIdx.y + blockIdx.x + 0 ];
        unsigned end_elem = elem_offsets[ wb * blockIdx.y + blockIdx.x + 1 ];
        for( int num_elem = beg_elem + threadIdx.x; num_elem < end_elem; num_elem += NB_THREADS_FOR_RASTER ) {
            int ind_elem = elem_data[ num_elem ];

            int n_0 = c_0[ ind_elem ];
            int n_1 = c_1[ ind_elem ];
            int n_2 = c_2[ ind_elem ];

            T x_0 = n_x[ n_0 ] - bx, y_0 = n_y[ n_0 ] - by, z_0 = n_z[ n_0 ], v_0 = -1;
            T x_1 = n_x[ n_1 ] - bx, y_1 = n_y[ n_1 ] - by, z_1 = n_z[ n_1 ], v_1 = -1;
            T x_2 = n_x[ n_2 ] - bx, y_2 = n_y[ n_2 ] - by, z_2 = n_z[ n_2 ], v_2 = -1;

            if ( n_v ) {
                T d = 1.0 / ( max_coloring_field - min_coloring_field );
                v_0 = ( n_v[ n_0 ] - min_coloring_field ) * d;
                v_1 = ( n_v[ n_1 ] - min_coloring_field ) * d;
                v_2 = ( n_v[ n_2 ] - min_coloring_field ) * d;
            }

            T3 normal = normalized( cross( T3( x_1, y_1, z_1 ) - T3( x_0, y_0, z_0 ), T3( x_2, y_2, z_2 ) - T3( x_0, y_0, z_0 ) ) );

            // sort points and values by y
            if ( y_0 > y_1 ) { swap( x_0, x_1 ); swap( y_0, y_1 ); swap( z_0, z_1 ); swap( v_0, v_1 ); }
            if ( y_0 > y_2 ) { swap( x_0, x_2 ); swap( y_0, y_2 ); swap( z_0, z_2 ); swap( v_0, v_2 ); }
            if ( y_1 > y_2 ) { swap( x_1, x_2 ); swap( y_1, y_2 ); swap( z_1, z_2 ); swap( v_1, v_2 ); }

            int dx_01 = x_1 - x_0, dy_01 = y_1 - y_0; float dz_01 = z_1 - z_0, dv_01 = v_1 - v_0;
            int dx_02 = x_2 - x_0, dy_02 = y_2 - y_0; float dz_02 = z_2 - z_0, dv_02 = v_2 - v_0;

            float cx_0 = dx_01 / float( dy_01 + not dy_01 ), cz_0 = dz_01 / float( dy_01 + not dy_01 ), cv_0 = dv_01 / float( dy_01 + not dy_01 );
            float cx_1 = dx_02 / float( dy_02 + not dy_02 ), cz_1 = dz_02 / float( dy_02 + not dy_02 ), cv_1 = dv_02 / float( dy_02 + not dy_02 );

            if ( cx_0 > cx_1 ) {
                swap( cx_0, cx_1 );
                swap( cz_0, cz_1 );
                swap( cv_0, cv_1 );
            }

            for( int y_b = max( 0, (int)y_0 ); y_b < min( NB_PIX_RASTER_BOX, (int)y_1 ); ++y_b ) {
                int xl_0 = x_0 + ( y_b - y_0 ) * cx_0;
                int xl_1 = x_0 + ( y_b - y_0 ) * cx_1;

                float zl_0 = z_0 + ( y_b - y_0 ) * cz_0;
                float zl_1 = z_0 + ( y_b - y_0 ) * cz_1;
                float vl_0 = v_0 + ( y_b - y_0 ) * cv_0;
                float vl_1 = v_0 + ( y_b - y_0 ) * cv_1;
                float c_z = xl_1 - xl_0; c_z = ( zl_1 - zl_0 ) / ( c_z + not c_z );
                float c_v = xl_1 - xl_0; c_v = ( vl_1 - vl_0 ) / ( c_v + not c_v );

                for( int x_b = max( 0, xl_0 ); x_b < min( NB_PIX_RASTER_BOX, xl_1 + 1 ); ++x_b ) {
                    float z_b = zl_0 + ( x_b - xl_0 ) * c_z;
                    unsigned z_bi = z_mul * ( z_b - z_min );
                    unsigned z_re = atomicMin( zznv_buffer + NB_PIX_RASTER_BOX * y_b + x_b, z_bi );
                    if ( z_re > z_bi ) {
                        float v = vl_0 + ( x_b - xl_0 ) * c_v;
                        float n = normal[ 2 ]; // dot( normal, trans.eye_dir( x_b + bx, y_b + by ) );
                        // if ( n < 0 ) continue;
                        rgba_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = shader( v, abs( n ) ); // + ( ( 150 * ( x_b == xl_0 or x_b == xl_1 ) ) << 8 );
                        nnnn_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = ind_elem;
                    }
                }
            }

            int dx_21 = x_1 - x_2, dy_21 = y_1 - y_2; float dz_21 = z_1 - z_2, dv_21 = v_1 - v_2;
            int dx_22 = x_0 - x_2, dy_22 = y_0 - y_2; float dz_22 = z_0 - z_2, dv_22 = v_0 - v_2;

            float cx_2 = dx_21 / float( dy_21 + not dy_21 ), cz_2 = dz_21 / float( dy_21 + not dy_21 ), cv_2 = dv_21 / float( dy_21 + not dy_21 );
            float cx_3 = dx_22 / float( dy_22 + not dy_22 ), cz_3 = dz_22 / float( dy_22 + not dy_22 ), cv_3 = dv_22 / float( dy_22 + not dy_22 );
            if ( cx_2 < cx_3 ) {
                swap( cx_2, cx_3 );
                swap( cz_2, cz_3 );
                swap( cv_2, cv_3 );
            }

            for( int y_b = max( 0, (int)y_1 ); y_b < min( NB_PIX_RASTER_BOX, (int)y_2 + 1 ); ++y_b ) {
                int xl_0 = x_2 + ( y_b - y_2 ) * cx_2;
                int xl_1 = x_2 + ( y_b - y_2 ) * cx_3;

                float zl_0 = z_2 + ( y_b - y_2 ) * cz_2;
                float zl_1 = z_2 + ( y_b - y_2 ) * cz_3;
                float vl_0 = v_2 + ( y_b - y_2 ) * cv_2;
                float vl_1 = v_2 + ( y_b - y_2 ) * cv_3;
                float c_z = xl_1 - xl_0; c_z = ( zl_1 - zl_0 ) / ( c_z + not c_z );
                float c_v = xl_1 - xl_0; c_v = ( vl_1 - vl_0 ) / ( c_v + not c_v );

                for( int x_b = max( 0, xl_0 ); x_b < min( NB_PIX_RASTER_BOX, xl_1 + 1 ); ++x_b ) {
                    float z_b = zl_0 + ( x_b - xl_0 ) * c_z;
                    unsigned z_bi = z_mul * ( z_b - z_min );
                    unsigned z_re = atomicMin( zznv_buffer + NB_PIX_RASTER_BOX * y_b + x_b, z_bi );
                    if ( z_re > z_bi ) {
                        float v = vl_0 + ( x_b - xl_0 ) * c_v;
                        float n = normal[ 2 ]; // dot( normal, trans.eye_dir( x_b + bx, y_b + by ) );
                        // if ( n < 0 ) continue;
                        rgba_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = shader( v, abs( n ) ); // + ( ( 150 * ( x_b == xl_0 or x_b == xl_1 ) ) << 8 );
                        nnnn_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = ind_elem;
                    }
                }
            }
        }
    }

    // declare local_z_min / max
    __shared__ unsigned local_z_min[ NB_THREADS_FOR_RASTER ];
    __shared__ unsigned local_z_max[ NB_THREADS_FOR_RASTER ];
    local_z_min[ threadIdx.x ] = 0xFFFFFFFF;
    local_z_max[ threadIdx.x ] = 0;

    // save result in img
    __syncthreads();
    for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
        int ox = i % NB_PIX_RASTER_BOX;
        int oy = i / NB_PIX_RASTER_BOX;
        int x = bx + ox;
        int y = by + oy;

        if ( x < w and y < h ) {
            rgba[ w * y + x ] = rgba_buffer[ i ] + 50 * ( oy == 0 or ox == 0 );
            zznv[ w * y + x ] = 0xFF000000 + zznv_buffer[ i ];
            nnnn[ w * y + x ] = nnnn_buffer[ i ];
            //
            if ( zznv_buffer[ i ] != 0xFFFFFFFF ) {
                local_z_min[ threadIdx.x ] = min( local_z_min[ threadIdx.x ], zznv_buffer[ i ] );
                local_z_max[ threadIdx.x ] = max( local_z_max[ threadIdx.x ], zznv_buffer[ i ] );
            }
        }
    }

    // save z_min z_max
    for( int m = NB_THREADS_FOR_RASTER / 2; m; m /= 2 ) {
        __syncthreads();
        if ( threadIdx.x < m ) {
            local_z_min[ threadIdx.x ] = min( local_z_min[ threadIdx.x ], local_z_min[ threadIdx.x + m ] );
            local_z_max[ threadIdx.x ] = max( local_z_max[ threadIdx.x ], local_z_max[ threadIdx.x + m ] );
        }
    }
}

void DisplayItem_BasicMesh::render_to( BitmapDisplay *display ) {
    ASSERT( mesh.pos->is_a_gpu(), "render works only with data on gpu" );

    // get various data
    int w = display->get_w();
    int h = display->get_h();
    int wb = iDivUp( w, NB_PIX_RASTER_BOX );
    int hb = iDivUp( h, NB_PIX_RASTER_BOX );
    int sb = wb * hb;

    DisplayTrans *trans = display->get_trans_gpu();

    ST nb_types = 0;
    CSC(( hipMemcpy( &nb_types, &mesh->elem_groups.size_, sizeof( ST ), hipMemcpyDeviceToHost ) ));

    ST nb_nodes = 0;
    CSC(( hipMemcpy( &nb_nodes, &mesh->pos_nodes.data_, sizeof( ST ), hipMemcpyDeviceToHost ) ));
    CSC(( hipMemcpy( &nb_nodes, &reinterpret_cast<const BasicVecRef<T> *>( nb_nodes )->size_, sizeof( ST ), hipMemcpyDeviceToHost ) ));
    for( int d = 0; d < 3; ++d )
        if ( not proj[ d ] )
            hipMalloc( proj + d, sizeof( T ) * nb_nodes );

    // make projection (integer values, stored in floating point numbers)
    CSC(( make_proj_kernel<<<iDivUp(nb_nodes,NB_THREADS_FOR_PROJ),NB_THREADS_FOR_PROJ>>>(
            proj[ 0 ], proj[ 1 ], proj[ 2 ], trans, w, h, mesh.ptr() ) ));

    // fill elem_count
    unsigned *elem_count = get_elem_count_gpu_ptr( nb_types, sb );
    CSC(( make_elem_count_kernel<<<NB_BLOCKS_FOR_ELEM_COUNT,128>>>( elem_count, wb, hb, sb, mesh.ptr(), proj[ 0 ], proj[ 1 ] ) ));
    CSC(( make_sum_elem_count_kernel<<<Metil::iDivUp(sb,64),64>>>( elem_count, wb, hb, nb_types ) ));
    CSC(( make_off_elem_count_kernel<<<1,1>>>( elem_count, wb, hb, nb_types ) ));

    // find size of elem_data list
    unsigned tot_nb_elems;
    CSC(( hipMemcpy( &tot_nb_elems, elem_count + nb_types * rese_elem_count_for_one_group( sb ),
                      sizeof( unsigned ), hipMemcpyDeviceToHost ) ));


    // fill elem_data
    int *elem_data = get_elem_data_gpu_ptr( tot_nb_elems );
    CSC(( make_elem_data_kernel<<<NB_BLOCKS_FOR_ELEM_COUNT,128>>>( elem_count, elem_data, trans, wb, hb, sb, mesh.ptr(), proj[ 0 ], proj[ 1 ] ) ));

    // fill image
    dim3 grid_size_raster( wb, hb );
    CSC(( raster_gpu_kernel<<<grid_size_raster,NB_THREADS_FOR_RASTER>>>(
             display->img_rgba.get_gpu_ptr(),
             display->img_zznv.get_gpu_ptr(),
             display->img_nnnn.get_gpu_ptr(),
             wb, hb, sb, w, h,
             elem_count + sb * NB_BLOCKS_FOR_ELEM_COUNT, elem_data,
             mesh.ptr(), proj[ 0 ], proj[ 1 ], proj[ 2 ],
             display->p_min[ 2 ], 65534.0 / plus_one_if_eqz( display->p_max[ 2 ] - display->p_min[ 2 ] ), display->first_item(),
             min_coloring_field, max_coloring_field,
             num_coloring_field, dim_coloring_field
    ) ));
}

void DisplayItem_BasicMesh::render_to( VectorDisplay * ) {
    TODO;
}

END_METIL_NAMESPACE;
