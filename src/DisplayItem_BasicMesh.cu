#include "hip/hip_runtime.h"
#include "DisplayItem_BasicMesh.h"
#include "BitmapDisplay.h"

BEG_METIL_NAMESPACE;

DisplayItem_BasicMesh::DisplayItem_BasicMesh( Ps<BasicMesh_Compacted> mesh ) : mesh( mesh ) {
}

__global__
void DisplayItem_BasicMesh_render_kernel( unsigned *img, int w, int h, const BasicMesh_Compacted *mesh ) {
    for( int y = 0; y < h; ++y )
        for( int x = 0; x < w; ++x )
            img[ y * w + x ] = 0xFF000000; // + ( y * 255 / h );

    const BasicVecRef<float> &x = mesh->pos_nodes[ 0 ];
    const BasicVecRef<float> &y = mesh->pos_nodes[ 1 ];
    const BasicVecRef<float> &z = mesh->pos_nodes[ 2 ];
    for( int i = 0; i < x.size(); ++i ) {
        img[ int( y[ i ] ) * w + int( x[ i ] ) ] = 0xFFFFFFFF;
    }
}


void DisplayItem_BasicMesh::render_to( BitmapDisplay *display ) {
    ASSERT( mesh.pos.is_a_gpu(), "render works only with data on gpu" );
    DisplayItem_BasicMesh_render_kernel<<<1,1>>>( display->get_img_gpu_ptr(), display->get_w(), display->get_h(), mesh.ptr() );
}

void DisplayItem_BasicMesh::render_to( VectorDisplay * ) {
    // fill elem_count
    unsigned *elem_count = get_elem_count_GPU_ptr( mesh.nb_types() );
    int size_for_1t = ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb() + 1;
    Display::Trans *trans = get_trans_GPU_ptr();
    CSC(( make_elem_count_kernel<<<NB_BLOCKS_FOR_ELEM_COUNT,128>>>( elem_count, trans, wb(), hb(), sb(), w, h, m ) ));
}

END_METIL_NAMESPACE;
