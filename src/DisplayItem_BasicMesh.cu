#include "hip/hip_runtime.h"
#include "GetSurroundingBox_BasicMesh.h"
#include "DisplayItem_BasicMesh.h"
#include "BitmapDisplay.h"
#include "MathBasicVec.h"
#include "CudaMetil.h"

#define NB_PIX_RASTER_BOX 32
#define NB_BLOCKS_FOR_ELEM_COUNT 32
#define MAX_WH ( 1600 * 1200 )
#define NB_THREADS_FOR_RASTER 64

BEG_METIL_NAMESPACE;

DisplayItem_BasicMesh::DisplayItem_BasicMesh( Ps<BasicMesh_Compacted> mesh ) : mesh( mesh ) {
    size_elem_count = 0;
    elem_count_gpu = 0;

    size_elem_data = 0;
    elem_data_gpu = 0;
}

__inline__ ST rese_elem_count_for_one_group( int sb ) {
    return ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1 /*offset list needs n + 1 terms...*/;
}

unsigned *DisplayItem_BasicMesh::get_elem_count_gpu_ptr( ST nb_types, int sb ) {
    int needed_size = nb_types * rese_elem_count_for_one_group( sb ) + 1 /*size for total nb elements*/;
    if ( size_elem_count < needed_size ) {
        if ( elem_count_gpu )
            hipFree( elem_count_gpu );
        CSC( hipMalloc( &elem_count_gpu, sizeof( unsigned ) * needed_size ) );
        size_elem_count = needed_size;
    }
    return elem_count_gpu;
}

int *DisplayItem_BasicMesh::get_elem_data_gpu_ptr( ST needed_size ) {
    if ( size_elem_data < needed_size ) {
        needed_size = needed_size * 8 / 7;
        if ( elem_data_gpu )
            hipFree( elem_data_gpu );
        CSC( hipMalloc( &elem_data_gpu, sizeof( int ) * needed_size ) );
        size_elem_data = needed_size;
    }
    return elem_data_gpu;
}

void DisplayItem_BasicMesh::update_p_min_p_max( GenericDisplay *display, T3 &p_min, T3 &p_max ) {
    T3 t_min;
    T3 t_max;
    get_surrounding_box( t_min.ptr(), t_max.ptr(), mesh.ptr(), display->get_trans_gpu(), display->get_w(), display->get_h() );
    for( int i = 0; i < 3; ++i ) {
        p_min[ i ] = min( p_min[ i ], t_min[ i ] );
        p_max[ i ] = max( p_max[ i ], t_max[ i ] );
    }
}

__global__
void make_elem_count_kernel( unsigned *elem_count, DisplayTrans *trans_ptr, int wb, int hb, int w, int h, BasicMesh_Compacted *m ) {
    typedef BasicVec<float,3> T3;

    int sb = wb * hb;
    elem_count += blockIdx.x * sb;

    DisplayTrans::Buf trans = trans_ptr->make_buf( w, h );

    const float *n_x = m->pos_nodes[ 0 ].ptr();
    const float *n_y = m->pos_nodes[ 1 ].ptr();
    const float *n_z = m->pos_nodes[ 2 ].ptr();

    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        __shared__ unsigned local_count[ MAX_WH / NB_PIX_RASTER_BOX / NB_PIX_RASTER_BOX ];
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            local_count[ i ] = 0;
        __syncthreads(); //wait until local_count is ok

        int nb_elem = m->elem_groups[ num_type ].connec[ 0 ].size();
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        int be = ( blockIdx.x + 0 ) * nb_elem / gridDim.x;
        int ee = ( blockIdx.x + 1 ) * nb_elem / gridDim.x;

        for( int i = be + threadIdx.x; i < ee; i += blockDim.x ) {
            int n_0 = c_0[ i ];
            int n_1 = c_1[ i ];
            int n_2 = c_2[ i ];

            T3 B_0 = trans.proj( n_x[ n_0 ], n_y[ n_0 ], n_z[ n_0 ] );
            T3 B_1 = trans.proj( n_x[ n_1 ], n_y[ n_1 ], n_z[ n_1 ] );
            T3 B_2 = trans.proj( n_x[ n_2 ], n_y[ n_2 ], n_z[ n_2 ] );

            int x_m = int( min( min( B_0[ 0 ], B_1[ 0 ] ), B_2[ 0 ] ) ) / NB_PIX_RASTER_BOX;
            int x_M = int( max( max( B_0[ 0 ], B_1[ 0 ] ), B_2[ 0 ] ) ) / NB_PIX_RASTER_BOX;
            int y_m = int( min( min( B_0[ 1 ], B_1[ 1 ] ), B_2[ 1 ] ) ) / NB_PIX_RASTER_BOX;
            int y_M = int( max( max( B_0[ 1 ], B_1[ 1 ] ), B_2[ 1 ] ) ) / NB_PIX_RASTER_BOX;

            for( int yb = max( 0, y_m ); yb <= min( hb - 1, y_M ); ++yb )
                for( int xb = max( 0, x_m ); xb <= min( wb - 1, x_M ); ++xb )
                    atomicAdd( local_count + wb * yb + xb, 1 );
        }

        // copy local count in global memory
        __syncthreads();
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            elem_count[ i ] = local_count[ i ];
    }
}


__global__
void make_sum_elem_count_kernel( unsigned *elem_count, int wb, int hb, int nb_types ) {
    int sb = wb * hb;
    for( int num_type = 0; num_type < nb_types; ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        int n = blockIdx.x * blockDim.x + threadIdx.x;
        if ( n < sb ) {
            unsigned acc = 0;
            for( int i = n; i < sb * NB_BLOCKS_FOR_ELEM_COUNT; i += sb ) {
                unsigned val = elem_count[ i ];
                elem_count[ i ] = acc;
                acc += val;
            }
            elem_count[ sb * NB_BLOCKS_FOR_ELEM_COUNT + n ] = acc;
        }
    }
}

// elem_count will be a list of offset. total number of elements is stored at the end of elem_count
__global__
void make_off_elem_count_kernel( unsigned *elem_count, int wb, int hb, int nb_types ) {
    int sb = wb * hb;
    int offset = sb * NB_BLOCKS_FOR_ELEM_COUNT;

    unsigned acc = 0;
    for( int num_type = 0; num_type < nb_types; ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        for( int i = 0; i < sb; ++i ) {
            unsigned val = elem_count[ offset + i ];
            elem_count[ offset + i ] = acc;
            acc += val;
        }
        elem_count[ offset + sb ] = acc;
    }

    *elem_count = acc;
}

///
__global__
void make_elem_data_kernel( unsigned *elem_count, int *elem_data, const DisplayTrans *trans_ptr, int wb, int hb, int sb, int w, int h, BasicMesh_Compacted *m ) {
    typedef BasicVec<float,3> T3;
    __shared__ unsigned local_count[ MAX_WH / NB_PIX_RASTER_BOX / NB_PIX_RASTER_BOX ];

    DisplayTrans::Buf trans = trans_ptr->make_buf( w, h );

    const float *n_x = m->pos_nodes[ 0 ].ptr();
    const float *n_y = m->pos_nodes[ 1 ].ptr();
    const float *n_z = m->pos_nodes[ 2 ].ptr();

    // int size_for_1t = ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1;
    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_count += rese_elem_count_for_one_group( sb ) ) {
        __syncthreads();
        for( int i = threadIdx.x; i < sb; i += blockDim.x )
            local_count[ i ] = elem_count[ NB_BLOCKS_FOR_ELEM_COUNT * sb + i ] + elem_count[ blockIdx.x * sb + i ];

        int nb_elem = m->elem_groups[ num_type ].connec[ 0 ].size();
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        int be = ( blockIdx.x + 0 ) * nb_elem / gridDim.x;
        int ee = ( blockIdx.x + 1 ) * nb_elem / gridDim.x;

        __syncthreads();
        for( int i = be + threadIdx.x; i < ee; i += blockDim.x ) {
            int n_0 = c_0[ i ];
            int n_1 = c_1[ i ];
            int n_2 = c_2[ i ];

            T3 B_0 = trans.proj( n_x[ n_0 ], n_y[ n_0 ], n_z[ n_0 ] );
            T3 B_1 = trans.proj( n_x[ n_1 ], n_y[ n_1 ], n_z[ n_1 ] );
            T3 B_2 = trans.proj( n_x[ n_2 ], n_y[ n_2 ], n_z[ n_2 ] );

            int x_m = int( min( min( B_0[ 0 ], B_1[ 0 ] ), B_2[ 0 ] ) ) / NB_PIX_RASTER_BOX;
            int x_M = int( max( max( B_0[ 0 ], B_1[ 0 ] ), B_2[ 0 ] ) ) / NB_PIX_RASTER_BOX;
            int y_m = int( min( min( B_0[ 1 ], B_1[ 1 ] ), B_2[ 1 ] ) ) / NB_PIX_RASTER_BOX;
            int y_M = int( max( max( B_0[ 1 ], B_1[ 1 ] ), B_2[ 1 ] ) ) / NB_PIX_RASTER_BOX;

            for( int yb = max( 0, y_m ); yb <= min( hb - 1, y_M ); ++yb ) {
                for( int xb = max( 0, x_m ); xb <= min( wb - 1, x_M ); ++xb ) {
                    unsigned index = atomicAdd( local_count + wb * yb + xb, 1 );
                    elem_data[ index + 0 ] = i;
                }
            }
        }
    }
}

__global__
void DisplayItem_BasicMesh_render_kernel( unsigned *img, const unsigned *elem_count, int w, int h, int wb, int hb ) {
    int sb = wb * hb;
    elem_count += sb;
    for( int y = 0; y < h; ++y )
        for( int x = 0; x < w; ++x )
            img[ y * w + x ] = 0xFF000000 + elem_count[ y / NB_PIX_RASTER_BOX * wb + x / NB_PIX_RASTER_BOX ] * 25;
}

__global__
void init_raster_GPU_kernel( unsigned *res, int w, int h ) {
    unsigned *info = res + 3 * w * h;
    info[ 4 ] = 0xFFFFFFFF;
    info[ 5 ] = 0xFF000000;
}


__inline__
unsigned shader( float z_n ) {
    int r = z_n * 200;
    int g = r;
    int b = r;
    int a = 255;
    return ( r << 0 ) + ( g << 8 ) + ( b << 16 ) + ( a << 24 );
}

/// use 2 unsigned to store a T ( 2 because of tha alpha chanel that must be == 0xFF )
__inline__ void copy_T_as_I( unsigned *res, float src ) {
    int e, s = src < 0;
    unsigned m = abs( frexpf( src, &e ) * 0x01000000 );
    res[ 0 ] = 0xFF000000 + m;
    res[ 1 ] = 0xFF000000 + ( s << 16 ) + ( e + 32768 );
}

__global__
void raster_gpu_kernel( unsigned *res, const DisplayTrans *trans_ptr, int wb, int hb, int sb, int w, int h, const unsigned *elem_offsets, const int *elem_data, const BasicMesh_Compacted *m, float z_min, float z_max, bool first_item ) {
    typedef DisplayTrans::T3 T3;

    int bx = blockIdx.x * NB_PIX_RASTER_BOX;
    int by = blockIdx.y * NB_PIX_RASTER_BOX;

    unsigned *rgba = res + 0 * w * h;
    unsigned *zznv = res + 1 * w * h;
    unsigned *nnnn = res + 2 * w * h;
    unsigned *info = res + 3 * w * h;


    // initialization of the z buffer
    __shared__ unsigned rgba_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    __shared__ unsigned zznv_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    __shared__ unsigned nnnn_buffer[ NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX ];
    if ( first_item ) {
        for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
            rgba_buffer[ i ] = 0x00000000;
            zznv_buffer[ i ] = 0xFFFFFFFF;
            nnnn_buffer[ i ] = 0xFFFFFFFF;
        }
    } else {
        for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
            int x = bx + i % NB_PIX_RASTER_BOX;
            int y = by + i / NB_PIX_RASTER_BOX;
            if ( x < w and y < h ) {
                rgba_buffer[ i ] = rgba[ w * y + x ];
                zznv_buffer[ i ] = zznv[ w * y + x ];
                nnnn_buffer[ i ] = nnnn[ w * y + x ];
            }
        }
    }

    DisplayTrans::Buf trans = trans_ptr->make_buf( w, h );

    const float *n_x = m->pos_nodes[ 0 ].ptr();
    const float *n_y = m->pos_nodes[ 1 ].ptr();
    const float *n_z = m->pos_nodes[ 2 ].ptr();

    int size_for_1t = ( NB_BLOCKS_FOR_ELEM_COUNT + 1 ) * sb + 1;
    for( int num_type = 0; num_type < m->elem_groups.size(); ++num_type, elem_offsets += size_for_1t ) {
        const int *c_0 = m->elem_groups[ num_type ].connec[ 0 ].ptr();
        const int *c_1 = m->elem_groups[ num_type ].connec[ 1 ].ptr();
        const int *c_2 = m->elem_groups[ num_type ].connec[ 2 ].ptr();

        // swwep elem list
        __syncthreads(); //wait until z_buffer is ok
        unsigned beg_elem = elem_offsets[ wb * blockIdx.y + blockIdx.x + 0 ];
        unsigned end_elem = elem_offsets[ wb * blockIdx.y + blockIdx.x + 1 ];
        for( int num_elem = beg_elem + threadIdx.x; num_elem < end_elem; num_elem += NB_THREADS_FOR_RASTER ) {
            int ind_elem = elem_data[ num_elem ];

            int n_0 = c_0[ ind_elem ];
            int n_1 = c_1[ ind_elem ];
            int n_2 = c_2[ ind_elem ];

            T3 P_0( n_x[ n_0 ], n_y[ n_0 ], n_z[ n_0 ] );
            T3 P_1( n_x[ n_1 ], n_y[ n_1 ], n_z[ n_1 ] );
            T3 P_2( n_x[ n_2 ], n_y[ n_2 ], n_z[ n_2 ] );
            T3 normal = normalized( cross( P_1 - P_0, P_2 - P_0 ) );

            T3 B_0 = trans.proj( P_0 );
            T3 B_1 = trans.proj( P_1 );
            T3 B_2 = trans.proj( P_2 );

            swap_if( B_0, B_1, B_0[ 1 ] > B_1[ 1 ] );
            swap_if( B_0, B_2, B_0[ 1 ] > B_2[ 1 ] );
            swap_if( B_1, B_2, B_1[ 1 ] > B_2[ 1 ] );

            int x_0 = int( B_0[ 0 ] ) - bx, y_0 = int( B_0[ 1 ] ) - by;
            int x_1 = int( B_1[ 0 ] ) - bx, y_1 = int( B_1[ 1 ] ) - by;
            int x_2 = int( B_2[ 0 ] ) - bx, y_2 = int( B_2[ 1 ] ) - by;

            int dx_01 = x_1 - x_0, dy_01 = y_1 - y_0; float dz_01 = B_1[ 2 ] - B_0[ 2 ];
            int dx_02 = x_2 - x_0, dy_02 = y_2 - y_0; float dz_02 = B_2[ 2 ] - B_0[ 2 ];

            float cx_0 = dx_01 / float( dy_01 + not dy_01 ), cz_0 = dz_01 / float( dy_01 + not dy_01 );
            float cx_1 = dx_02 / float( dy_02 + not dy_02 ), cz_1 = dz_02 / float( dy_02 + not dy_02 );
            bool cond_0 = cx_0 > cx_1;
            swap_if( cx_0, cx_1, cond_0 );
            swap_if( cz_0, cz_1, cond_0 );

            for( int y_b = max( 0, y_0 ); y_b < min( NB_PIX_RASTER_BOX, y_1 ); ++y_b ) {
                int xl_0 = x_0 + ( y_b - y_0 ) * cx_0;
                int xl_1 = x_0 + ( y_b - y_0 ) * cx_1;

                float zl_0 = B_0[ 2 ] + ( y_b - y_0 ) * cz_0;
                float zl_1 = B_0[ 2 ] + ( y_b - y_0 ) * cz_1;
                float c_z = xl_1 - xl_0; c_z = ( zl_1 - zl_0 ) / ( c_z + not c_z );

                for( int x_b = max( 0, xl_0 ); x_b < min( NB_PIX_RASTER_BOX, xl_1 + 1 ); ++x_b ) {
                    float z_b = zl_0 + ( x_b - xl_0 ) * c_z;
                    unsigned z_bi = ( z_b - z_min ) * 65534 / ( z_max - z_min );
                    unsigned z_re = atomicMin( zznv_buffer + NB_PIX_RASTER_BOX * y_b + x_b, z_bi );
                    if ( z_re > z_bi ) {
                        float c = abs( dot( normal, trans.eye_dir( x_b + bx, y_b + by ) ) );
                        rgba_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = shader( c );
                        nnnn_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = ind_elem;
                    }
                }
            }

            int dx_21 = x_1 - x_2, dy_21 = y_1 - y_2, dz_21 = B_1[ 2 ] - B_2[ 2 ];
            int dx_22 = x_0 - x_2, dy_22 = y_0 - y_2, dz_22 = B_0[ 2 ] - B_2[ 2 ];

            float cx_2 = dx_21 / float( dy_21 + not dy_21 ), cz_2 = dz_21 / float( dy_21 + not dy_21 );
            float cx_3 = dx_22 / float( dy_22 + not dy_22 ), cz_3 = dz_22 / float( dy_22 + not dy_22 );
            bool cond_1 = cx_2 < cx_3;
            swap_if( cx_2, cx_3, cond_1 );
            swap_if( cz_2, cz_3, cond_1 );

            for( int y_b = max( 0, y_1 ); y_b < min( NB_PIX_RASTER_BOX, y_2 + 1 ); ++y_b ) {
                int xl_0 = x_2 + ( y_b - y_2 ) * cx_2;
                int xl_1 = x_2 + ( y_b - y_2 ) * cx_3;

                float zl_0 = B_2[ 2 ] + ( y_b - y_2 ) * cz_2;
                float zl_1 = B_2[ 2 ] + ( y_b - y_2 ) * cz_3;
                float c_z = xl_1 - xl_0; c_z = ( zl_1 - zl_0 ) / ( c_z + not c_z );

                for( int x_b = max( 0, xl_0 ); x_b < min( NB_PIX_RASTER_BOX, xl_1 + 1 ); ++x_b ) {
                    float z_b = zl_0 + ( x_b - xl_0 ) * c_z;
                    unsigned z_bi = ( z_b - z_min ) * 65534 / ( z_max - z_min );
                    unsigned z_re = atomicMin( zznv_buffer + NB_PIX_RASTER_BOX * y_b + x_b, z_bi );
                    if ( z_re > z_bi ) {
                        float c = abs( dot( normal, trans.eye_dir( x_b + bx, y_b + by ) ) );
                        rgba_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = shader( c );
                        nnnn_buffer[ NB_PIX_RASTER_BOX * y_b + x_b ] = ind_elem;
                    }
                }
            }
        }
    }

    // declare local_z_min / max
    __shared__ unsigned local_z_min[ NB_THREADS_FOR_RASTER ];
    __shared__ unsigned local_z_max[ NB_THREADS_FOR_RASTER ];
    local_z_min[ threadIdx.x ] = 0xFFFFFFFF;
    local_z_max[ threadIdx.x ] = 0;

    // save result in img
    __syncthreads();
    for( int i = threadIdx.x; i < NB_PIX_RASTER_BOX * NB_PIX_RASTER_BOX; i += NB_THREADS_FOR_RASTER ) {
        int x = bx + i % NB_PIX_RASTER_BOX;
        int y = by + i / NB_PIX_RASTER_BOX;

        if ( x < w and y < h ) {
            rgba[ w * y + x ] = rgba_buffer[ i ];
            zznv[ w * y + x ] = 0xFF000000 + zznv_buffer[ i ];
            nnnn[ w * y + x ] = 0xFF000000 + nnnn_buffer[ i ];
            //
            if ( zznv_buffer[ i ] != 0xFFFFFFFF ) {
                local_z_min[ threadIdx.x ] = min( local_z_min[ threadIdx.x ], zznv_buffer[ i ] );
                local_z_max[ threadIdx.x ] = max( local_z_max[ threadIdx.x ], zznv_buffer[ i ] );
            }
        }
    }

    // save z_min z_max
    for( int m = NB_THREADS_FOR_RASTER / 2; m; m /= 2 ) {
        __syncthreads();
        if ( threadIdx.x < m ) {
            local_z_min[ threadIdx.x ] = min( local_z_min[ threadIdx.x ], local_z_min[ threadIdx.x + m ] );
            local_z_max[ threadIdx.x ] = max( local_z_max[ threadIdx.x ], local_z_max[ threadIdx.x + m ] );
        }
    }

    if ( threadIdx.x == 0 ) {
        if ( blockIdx.x == 0 ) {
            copy_T_as_I( info + 0, z_min );
            copy_T_as_I( info + 2, z_max );
        }
        if ( *local_z_min != 0xFFFFFFFF )
            atomicMin( info + 4, 0xFF000000 + ( *local_z_min/* >> 8*/ ) );
        if ( *local_z_max != 0x0 )
            atomicMax( info + 5, 0xFF000000 + ( *local_z_max/* >> 8*/ ) );
    }
}


void DisplayItem_BasicMesh::render_to( BitmapDisplay *display ) {
    ASSERT( mesh.pos.is_a_gpu(), "render works only with data on gpu" );

    int w = display->get_w();
    int h = display->get_h();
    int wb = iDivUp( w, NB_PIX_RASTER_BOX );
    int hb = iDivUp( h, NB_PIX_RASTER_BOX );
    int sb = wb * hb;

    ST nb_types;
    hipMemcpy( &nb_types, &mesh->elem_groups.size_, sizeof( ST ), hipMemcpyDeviceToHost );

    // fill elem_count
    unsigned *elem_count = get_elem_count_gpu_ptr( nb_types, sb );
    DisplayTrans *trans = display->get_trans_gpu();
    CSC(( make_elem_count_kernel<<<NB_BLOCKS_FOR_ELEM_COUNT,128>>>( elem_count, trans, wb, hb, w, h, mesh.ptr() ) ));
    CSC(( make_sum_elem_count_kernel<<<Metil::iDivUp(sb,64),64>>>( elem_count, wb, hb, nb_types ) ));
    CSC(( make_off_elem_count_kernel<<<1,1>>>( elem_count, wb, hb, nb_types ) ));

    // find size of elem_data list
    unsigned tot_nb_elems;
    CSC(( hipMemcpy( &tot_nb_elems, elem_count + nb_types * rese_elem_count_for_one_group( sb ),
                      sizeof( unsigned ), hipMemcpyDeviceToHost ) ));


    // fill elem_data
    int *elem_data = get_elem_data_gpu_ptr( tot_nb_elems );
    CSC(( make_elem_data_kernel<<<NB_BLOCKS_FOR_ELEM_COUNT,128>>>( elem_count, elem_data, trans, wb, hb, sb, w, h, mesh.ptr() ) ));

    // init info structure at the end of the image
    init_raster_GPU_kernel<<<1,1>>>( display->get_img_gpu_ptr(), w, h );

    // fill image
    dim3 grid_size_raster( wb, hb );
    CSC(( raster_gpu_kernel<<<grid_size_raster,NB_THREADS_FOR_RASTER>>>(
            display->get_img_gpu_ptr(), trans, wb, hb, sb, w, h,
            elem_count + sb * NB_BLOCKS_FOR_ELEM_COUNT,
            elem_data, mesh.ptr(), display->p_min[ 2 ], display->p_max[ 2 ], display->first_item()
    ) ));
}

void DisplayItem_BasicMesh::render_to( VectorDisplay * ) {
    TODO;
}

END_METIL_NAMESPACE;
