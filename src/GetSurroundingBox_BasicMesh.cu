#include "hip/hip_runtime.h"
#include "GetSurroundingBox_BasicMesh.h"
#include "CudaMetil.h"
#include "String.h"

#define NB_THREADS_MIN_MAX 64
#define NB_BLOCKS_MIN_MAX  64

BEG_METIL_NAMESPACE;

__global__
void get_min_max_transformation_kernel_0( float *res, const BasicMesh_Compacted *m, const DisplayTrans *trans_ptr, int w, int h ) {
    typedef DisplayTrans::T  T;
    typedef DisplayTrans::T3 T3;

    float *min_list[ 3 ], *max_list[ 3 ];
    for( int d = 0; d < 3; ++d ) {
        min_list[ d ] = res + ( 2 * d + 0 ) * NB_BLOCKS_MIN_MAX;
        max_list[ d ] = res + ( 2 * d + 1 ) * NB_BLOCKS_MIN_MAX;
    }

    int nb_nodes = m->pos_nodes[ 0 ].size();
    const float *n_x = m->pos_nodes[ 0 ].ptr();
    const float *n_y = m->pos_nodes[ 1 ].ptr();
    const float *n_z = m->pos_nodes[ 2 ].ptr();

    __shared__ float local_min[ 3 * NB_THREADS_MIN_MAX ];
    __shared__ float local_max[ 3 * NB_THREADS_MIN_MAX ];

    for( int d = 0; d < 3; ++d ) {
        local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = + HUGE;
        local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = - HUGE;
    }

    if ( trans_ptr ) {
        DisplayTrans::Buf trans = trans_ptr->make_buf( w, h );
        for( int index = blockIdx.x * NB_THREADS_MIN_MAX + threadIdx.x; index < nb_nodes; index += NB_THREADS_MIN_MAX ) {
            T3 P = trans.proj( n_x[ index ], n_y[ index ], n_z[ index ] );
            for( int d = 0; d < 3; ++d ) {
                local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = min( local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ], P[ d ] );
                local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = max( local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ], P[ d ] );
            }
        }
    } else {
        for( int index = blockIdx.x * NB_THREADS_MIN_MAX + threadIdx.x;index < nb_nodes; index += NB_THREADS_MIN_MAX ) {
            T3 P( n_x[ index ], n_y[ index ], n_z[ index ] );
            for( int d = 0; d < 3; ++d ) {
                local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = min( local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ], P[ d ] );
                local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = max( local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ], P[ d ] );
            }
        }
    }

    // reduction
    for( int m = NB_THREADS_MIN_MAX / 2; m; m /= 2 ) {
        syncthreads();
        if ( threadIdx.x < m ) {
            for( int d = 0; d < 3; ++d ) {
                local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = min( local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x ], local_min[ d * NB_THREADS_MIN_MAX + threadIdx.x + m ] );
                local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ] = max( local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x ], local_max[ d * NB_THREADS_MIN_MAX + threadIdx.x + m ] );
            }
        }
    }

    // -> mi and ma
    syncthreads();
    for( int d = 0; d < 3; ++d ) {
        min_list[ d ][ blockIdx.x ] = local_min[ d * NB_THREADS_MIN_MAX ];
        max_list[ d ][ blockIdx.x ] = local_max[ d * NB_THREADS_MIN_MAX ];
    }
}

__global__
void get_min_max_transformation_kernel_1( float *res ) {
    float *min_list[ 3 ], *max_list[ 3 ];
    for( int d = 0; d < 3; ++d ) {
        min_list[ d ] = res + ( 2 * d + 0 ) * NB_BLOCKS_MIN_MAX;
        max_list[ d ] = res + ( 2 * d + 1 ) * NB_BLOCKS_MIN_MAX;
    }

    for( int m = NB_BLOCKS_MIN_MAX / 2; m; m /= 2 ) {
        syncthreads();
        for( int i = threadIdx.x; i < m; i += NB_BLOCKS_MIN_MAX ) {
            for( int c = 0; c < 3; ++c ) {
                min_list[ c ][ i ] = min( min_list[ c ][ i ], min_list[ c ][ i + m ] );
                max_list[ c ][ i ] = max( max_list[ c ][ i ], max_list[ c ][ i + m ] );
            }
        }
    }
}

void get_surrounding_box( float *mi, float *ma, BasicMesh_Compacted *m, DisplayTrans *trans, int w, int h ) {
    float *res;
    CSC(( hipMalloc( &res, 3 * 2 * NB_BLOCKS_MIN_MAX * sizeof( float ) ) ));

    CSC(( get_min_max_transformation_kernel_0<<<NB_BLOCKS_MIN_MAX,NB_THREADS_MIN_MAX>>>( res, m, trans, w, h ) ));
    CSC(( get_min_max_transformation_kernel_1<<<                1, NB_BLOCKS_MIN_MAX>>>( res ) ));

    for( int d = 0; d < 3; ++d ) {
        CSC(( hipMemcpy( mi + d, res + ( 2 * d + 0 ) * NB_BLOCKS_MIN_MAX, sizeof( float ), hipMemcpyDeviceToHost ) ));
        CSC(( hipMemcpy( ma + d, res + ( 2 * d + 1 ) * NB_BLOCKS_MIN_MAX, sizeof( float ), hipMemcpyDeviceToHost ) ));
    }
}


END_METIL_NAMESPACE;
