#include "hip/hip_runtime.h"
#include "BasicMesh_Compacted.h"

BEG_METIL_NAMESPACE

void BasicMesh_Compacted::update_ptr_cpu_load( ST off ) {
    (char *&)pos_nodes.data_ += off;
    for( ST i = 0; i < pos_nodes.size_; ++i ) {
        (char *&)pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)elem_groups.data_ += off;
    for( ST i = 0; i < elem_groups.size_; ++i ) {
        (char *&)elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
    }
    (char *&)node_to_elem.data_ += off;
    for( ST i = 0; i < node_to_elem.size_; ++i ) {
        (char *&)node_to_elem.data_[ i ].data_ += off;
        for( ST j = 0; j < node_to_elem.data_[ i ].size_; ++j ) {
        }
    }
}

__global__
void BasicMesh_Compacted__update_ptr_gpu_load( BasicMesh_Compacted *obj, ST off ) {
    (char *&)obj->pos_nodes.data_ += off;
    for( ST i = 0; i < obj->pos_nodes.size_; ++i ) {
        (char *&)obj->pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->elem_groups.data_ += off;
    for( ST i = 0; i < obj->elem_groups.size_; ++i ) {
        (char *&)obj->elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < obj->elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
    }
    (char *&)obj->node_to_elem.data_ += off;
    for( ST i = 0; i < obj->node_to_elem.size_; ++i ) {
        (char *&)obj->node_to_elem.data_[ i ].data_ += off;
        for( ST j = 0; j < obj->node_to_elem.data_[ i ].size_; ++j ) {
        }
    }
}

void BasicMesh_Compacted::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < pos_nodes.size_; ++i ) {
        (char *&)pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)pos_nodes.data_ += off;
    for( ST i = 0; i < elem_groups.size_; ++i ) {
        for( ST j = 0; j < elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)elem_groups.data_[ i ].connec.data_ += off;
    }
    (char *&)elem_groups.data_ += off;
    for( ST i = 0; i < node_to_elem.size_; ++i ) {
        for( ST j = 0; j < node_to_elem.data_[ i ].size_; ++j ) {
        }
        (char *&)node_to_elem.data_[ i ].data_ += off;
    }
    (char *&)node_to_elem.data_ += off;
}

__global__
void BasicMesh_Compacted__update_ptr_gpu_save( BasicMesh_Compacted *obj, ST off ) {
    for( ST i = 0; i < obj->pos_nodes.size_; ++i ) {
        (char *&)obj->pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->pos_nodes.data_ += off;
    for( ST i = 0; i < obj->elem_groups.size_; ++i ) {
        for( ST j = 0; j < obj->elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)obj->elem_groups.data_[ i ].connec.data_ += off;
    }
    (char *&)obj->elem_groups.data_ += off;
    for( ST i = 0; i < obj->node_to_elem.size_; ++i ) {
        for( ST j = 0; j < obj->node_to_elem.data_[ i ].size_; ++j ) {
        }
        (char *&)obj->node_to_elem.data_[ i ].data_ += off;
    }
    (char *&)obj->node_to_elem.data_ += off;
}

void BasicMesh_Compacted::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::update_ptr_cpu_load( ST off ) {
    (char *&)connec.data_ += off;
    for( ST i = 0; i < connec.size_; ++i ) {
        (char *&)connec.data_[ i ].data_ += off;
    }
}

__global__
void BasicMesh_Compacted__ElemGroup__update_ptr_gpu_load( BasicMesh_Compacted::ElemGroup *obj, ST off ) {
    (char *&)obj->connec.data_ += off;
    for( ST i = 0; i < obj->connec.size_; ++i ) {
        (char *&)obj->connec.data_[ i ].data_ += off;
    }
}

void BasicMesh_Compacted::ElemGroup::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__ElemGroup__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < connec.size_; ++i ) {
        (char *&)connec.data_[ i ].data_ += off;
    }
    (char *&)connec.data_ += off;
}

__global__
void BasicMesh_Compacted__ElemGroup__update_ptr_gpu_save( BasicMesh_Compacted::ElemGroup *obj, ST off ) {
    for( ST i = 0; i < obj->connec.size_; ++i ) {
        (char *&)obj->connec.data_[ i ].data_ += off;
    }
    (char *&)obj->connec.data_ += off;
}

void BasicMesh_Compacted::ElemGroup::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__ElemGroup__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_cpu_load( ST off ) {
}

__global__
void BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_load( BasicMesh_Compacted::NodeToElemItem *obj, ST off ) {
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_cpu_save( ST off ) {
}

__global__
void BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_save( BasicMesh_Compacted::NodeToElemItem *obj, ST off ) {
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_save<<<1,1>>>( this, off );
}


END_METIL_NAMESPACE
