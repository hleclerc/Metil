#include "hip/hip_runtime.h"
#include "BasicMesh_Compacted.h"

BEG_METIL_NAMESPACE

void BasicMesh_Compacted::update_ptr_cpu_load( ST off ) {
    (char *&)pos_nodes.data_ += off;
    for( ST i = 0; i < pos_nodes.size_; ++i ) {
        (char *&)pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)elem_groups.data_ += off;
    for( ST i = 0; i < elem_groups.size_; ++i ) {
        (char *&)elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)elem_groups.data_[ i ].fields.data_ += off;
        for( ST j = 0; j < elem_groups.data_[ i ].fields.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].fields.data_[ j ].name.data_ += off;
            (char *&)elem_groups.data_[ i ].fields.data_[ j ].data.data_ += off;
            for( ST k = 0; k < elem_groups.data_[ i ].fields.data_[ j ].data.size_; ++k ) {
                (char *&)elem_groups.data_[ i ].fields.data_[ j ].data.data_[ k ].data_ += off;
            }
        }
    }
    (char *&)node_to_elem.data_ += off;
    for( ST i = 0; i < node_to_elem.size_; ++i ) {
        (char *&)node_to_elem.data_[ i ].data_ += off;
        for( ST j = 0; j < node_to_elem.data_[ i ].size_; ++j ) {
        }
    }
    (char *&)nodal_fields.data_ += off;
    for( ST i = 0; i < nodal_fields.size_; ++i ) {
        (char *&)nodal_fields.data_[ i ].name.data_ += off;
        (char *&)nodal_fields.data_[ i ].data.data_ += off;
        for( ST j = 0; j < nodal_fields.data_[ i ].data.size_; ++j ) {
            (char *&)nodal_fields.data_[ i ].data.data_[ j ].data_ += off;
        }
    }
}

__global__
void BasicMesh_Compacted__update_ptr_gpu_load( BasicMesh_Compacted *obj, ST off ) {
    (char *&)obj->pos_nodes.data_ += off;
    for( ST i = 0; i < obj->pos_nodes.size_; ++i ) {
        (char *&)obj->pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->elem_groups.data_ += off;
    for( ST i = 0; i < obj->elem_groups.size_; ++i ) {
        (char *&)obj->elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < obj->elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)obj->elem_groups.data_[ i ].fields.data_ += off;
        for( ST j = 0; j < obj->elem_groups.data_[ i ].fields.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].name.data_ += off;
            (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].data.data_ += off;
            for( ST k = 0; k < obj->elem_groups.data_[ i ].fields.data_[ j ].data.size_; ++k ) {
                (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].data.data_[ k ].data_ += off;
            }
        }
    }
    (char *&)obj->node_to_elem.data_ += off;
    for( ST i = 0; i < obj->node_to_elem.size_; ++i ) {
        (char *&)obj->node_to_elem.data_[ i ].data_ += off;
        for( ST j = 0; j < obj->node_to_elem.data_[ i ].size_; ++j ) {
        }
    }
    (char *&)obj->nodal_fields.data_ += off;
    for( ST i = 0; i < obj->nodal_fields.size_; ++i ) {
        (char *&)obj->nodal_fields.data_[ i ].name.data_ += off;
        (char *&)obj->nodal_fields.data_[ i ].data.data_ += off;
        for( ST j = 0; j < obj->nodal_fields.data_[ i ].data.size_; ++j ) {
            (char *&)obj->nodal_fields.data_[ i ].data.data_[ j ].data_ += off;
        }
    }
}

void BasicMesh_Compacted::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < pos_nodes.size_; ++i ) {
        (char *&)pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)pos_nodes.data_ += off;
    for( ST i = 0; i < elem_groups.size_; ++i ) {
        for( ST j = 0; j < elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < elem_groups.data_[ i ].fields.size_; ++j ) {
            (char *&)elem_groups.data_[ i ].fields.data_[ j ].name.data_ += off;
            for( ST k = 0; k < elem_groups.data_[ i ].fields.data_[ j ].data.size_; ++k ) {
                (char *&)elem_groups.data_[ i ].fields.data_[ j ].data.data_[ k ].data_ += off;
            }
            (char *&)elem_groups.data_[ i ].fields.data_[ j ].data.data_ += off;
        }
        (char *&)elem_groups.data_[ i ].fields.data_ += off;
    }
    (char *&)elem_groups.data_ += off;
    for( ST i = 0; i < node_to_elem.size_; ++i ) {
        for( ST j = 0; j < node_to_elem.data_[ i ].size_; ++j ) {
        }
        (char *&)node_to_elem.data_[ i ].data_ += off;
    }
    (char *&)node_to_elem.data_ += off;
    for( ST i = 0; i < nodal_fields.size_; ++i ) {
        (char *&)nodal_fields.data_[ i ].name.data_ += off;
        for( ST j = 0; j < nodal_fields.data_[ i ].data.size_; ++j ) {
            (char *&)nodal_fields.data_[ i ].data.data_[ j ].data_ += off;
        }
        (char *&)nodal_fields.data_[ i ].data.data_ += off;
    }
    (char *&)nodal_fields.data_ += off;
}

__global__
void BasicMesh_Compacted__update_ptr_gpu_save( BasicMesh_Compacted *obj, ST off ) {
    for( ST i = 0; i < obj->pos_nodes.size_; ++i ) {
        (char *&)obj->pos_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->pos_nodes.data_ += off;
    for( ST i = 0; i < obj->elem_groups.size_; ++i ) {
        for( ST j = 0; j < obj->elem_groups.data_[ i ].connec.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].connec.data_[ j ].data_ += off;
        }
        (char *&)obj->elem_groups.data_[ i ].connec.data_ += off;
        for( ST j = 0; j < obj->elem_groups.data_[ i ].fields.size_; ++j ) {
            (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].name.data_ += off;
            for( ST k = 0; k < obj->elem_groups.data_[ i ].fields.data_[ j ].data.size_; ++k ) {
                (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].data.data_[ k ].data_ += off;
            }
            (char *&)obj->elem_groups.data_[ i ].fields.data_[ j ].data.data_ += off;
        }
        (char *&)obj->elem_groups.data_[ i ].fields.data_ += off;
    }
    (char *&)obj->elem_groups.data_ += off;
    for( ST i = 0; i < obj->node_to_elem.size_; ++i ) {
        for( ST j = 0; j < obj->node_to_elem.data_[ i ].size_; ++j ) {
        }
        (char *&)obj->node_to_elem.data_[ i ].data_ += off;
    }
    (char *&)obj->node_to_elem.data_ += off;
    for( ST i = 0; i < obj->nodal_fields.size_; ++i ) {
        (char *&)obj->nodal_fields.data_[ i ].name.data_ += off;
        for( ST j = 0; j < obj->nodal_fields.data_[ i ].data.size_; ++j ) {
            (char *&)obj->nodal_fields.data_[ i ].data.data_[ j ].data_ += off;
        }
        (char *&)obj->nodal_fields.data_[ i ].data.data_ += off;
    }
    (char *&)obj->nodal_fields.data_ += off;
}

void BasicMesh_Compacted::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::update_ptr_cpu_load( ST off ) {
    (char *&)connec.data_ += off;
    for( ST i = 0; i < connec.size_; ++i ) {
        (char *&)connec.data_[ i ].data_ += off;
    }
    (char *&)fields.data_ += off;
    for( ST i = 0; i < fields.size_; ++i ) {
        (char *&)fields.data_[ i ].name.data_ += off;
        (char *&)fields.data_[ i ].data.data_ += off;
        for( ST j = 0; j < fields.data_[ i ].data.size_; ++j ) {
            (char *&)fields.data_[ i ].data.data_[ j ].data_ += off;
        }
    }
}

__global__
void BasicMesh_Compacted__ElemGroup__update_ptr_gpu_load( BasicMesh_Compacted::ElemGroup *obj, ST off ) {
    (char *&)obj->connec.data_ += off;
    for( ST i = 0; i < obj->connec.size_; ++i ) {
        (char *&)obj->connec.data_[ i ].data_ += off;
    }
    (char *&)obj->fields.data_ += off;
    for( ST i = 0; i < obj->fields.size_; ++i ) {
        (char *&)obj->fields.data_[ i ].name.data_ += off;
        (char *&)obj->fields.data_[ i ].data.data_ += off;
        for( ST j = 0; j < obj->fields.data_[ i ].data.size_; ++j ) {
            (char *&)obj->fields.data_[ i ].data.data_[ j ].data_ += off;
        }
    }
}

void BasicMesh_Compacted::ElemGroup::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__ElemGroup__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < connec.size_; ++i ) {
        (char *&)connec.data_[ i ].data_ += off;
    }
    (char *&)connec.data_ += off;
    for( ST i = 0; i < fields.size_; ++i ) {
        (char *&)fields.data_[ i ].name.data_ += off;
        for( ST j = 0; j < fields.data_[ i ].data.size_; ++j ) {
            (char *&)fields.data_[ i ].data.data_[ j ].data_ += off;
        }
        (char *&)fields.data_[ i ].data.data_ += off;
    }
    (char *&)fields.data_ += off;
}

__global__
void BasicMesh_Compacted__ElemGroup__update_ptr_gpu_save( BasicMesh_Compacted::ElemGroup *obj, ST off ) {
    for( ST i = 0; i < obj->connec.size_; ++i ) {
        (char *&)obj->connec.data_[ i ].data_ += off;
    }
    (char *&)obj->connec.data_ += off;
    for( ST i = 0; i < obj->fields.size_; ++i ) {
        (char *&)obj->fields.data_[ i ].name.data_ += off;
        for( ST j = 0; j < obj->fields.data_[ i ].data.size_; ++j ) {
            (char *&)obj->fields.data_[ i ].data.data_[ j ].data_ += off;
        }
        (char *&)obj->fields.data_[ i ].data.data_ += off;
    }
    (char *&)obj->fields.data_ += off;
}

void BasicMesh_Compacted::ElemGroup::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__ElemGroup__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::Field::update_ptr_cpu_load( ST off ) {
    (char *&)name.data_ += off;
    (char *&)data.data_ += off;
    for( ST i = 0; i < data.size_; ++i ) {
        (char *&)data.data_[ i ].data_ += off;
    }
}

__global__
void BasicMesh_Compacted__ElemGroup__Field__update_ptr_gpu_load( BasicMesh_Compacted::ElemGroup::Field *obj, ST off ) {
    (char *&)obj->name.data_ += off;
    (char *&)obj->data.data_ += off;
    for( ST i = 0; i < obj->data.size_; ++i ) {
        (char *&)obj->data.data_[ i ].data_ += off;
    }
}

void BasicMesh_Compacted::ElemGroup::Field::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__ElemGroup__Field__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::ElemGroup::Field::update_ptr_cpu_save( ST off ) {
    (char *&)name.data_ += off;
    for( ST i = 0; i < data.size_; ++i ) {
        (char *&)data.data_[ i ].data_ += off;
    }
    (char *&)data.data_ += off;
}

__global__
void BasicMesh_Compacted__ElemGroup__Field__update_ptr_gpu_save( BasicMesh_Compacted::ElemGroup::Field *obj, ST off ) {
    (char *&)obj->name.data_ += off;
    for( ST i = 0; i < obj->data.size_; ++i ) {
        (char *&)obj->data.data_[ i ].data_ += off;
    }
    (char *&)obj->data.data_ += off;
}

void BasicMesh_Compacted::ElemGroup::Field::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__ElemGroup__Field__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_cpu_load( ST off ) {
}

__global__
void BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_load( BasicMesh_Compacted::NodeToElemItem *obj, ST off ) {
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_cpu_save( ST off ) {
}

__global__
void BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_save( BasicMesh_Compacted::NodeToElemItem *obj, ST off ) {
}

void BasicMesh_Compacted::NodeToElemItem::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__NodeToElemItem__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::Field::update_ptr_cpu_load( ST off ) {
    (char *&)name.data_ += off;
    (char *&)data.data_ += off;
    for( ST i = 0; i < data.size_; ++i ) {
        (char *&)data.data_[ i ].data_ += off;
    }
}

__global__
void BasicMesh_Compacted__Field__update_ptr_gpu_load( BasicMesh_Compacted::Field *obj, ST off ) {
    (char *&)obj->name.data_ += off;
    (char *&)obj->data.data_ += off;
    for( ST i = 0; i < obj->data.size_; ++i ) {
        (char *&)obj->data.data_[ i ].data_ += off;
    }
}

void BasicMesh_Compacted::Field::update_ptr_gpu_load( ST off ) {
    BasicMesh_Compacted__Field__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void BasicMesh_Compacted::Field::update_ptr_cpu_save( ST off ) {
    (char *&)name.data_ += off;
    for( ST i = 0; i < data.size_; ++i ) {
        (char *&)data.data_[ i ].data_ += off;
    }
    (char *&)data.data_ += off;
}

__global__
void BasicMesh_Compacted__Field__update_ptr_gpu_save( BasicMesh_Compacted::Field *obj, ST off ) {
    (char *&)obj->name.data_ += off;
    for( ST i = 0; i < obj->data.size_; ++i ) {
        (char *&)obj->data.data_[ i ].data_ += off;
    }
    (char *&)obj->data.data_ += off;
}

void BasicMesh_Compacted::Field::update_ptr_gpu_save( ST off ) {
    BasicMesh_Compacted__Field__update_ptr_gpu_save<<<1,1>>>( this, off );
}


END_METIL_NAMESPACE
